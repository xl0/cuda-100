#include <stdio.h>
#include <hip/hip_runtime.h>


#ifndef N_THREADS
    #define N_THREADS 512
#endif


__global__ void vecAddKernel(float *a, float *b, float *c, uint n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        c[i] = a[i]+ b[i];
    }

}


void vecAdd_f32(float *A, float *B, float *C, uint n) {
    float *A_d, *B_d, *C_d;
    int size = n * sizeof(float);

    hipMalloc((void **) &A_d, size);
    hipMalloc((void **) &B_d, size);
    hipMalloc((void **) &C_d, size);


    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    vecAddKernel <<<(n + N_THREADS - 1) / N_THREADS, N_THREADS>>>(A_d, B_d, C_d, n);

    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}



void add_vectors_cpu(float *a, float *b, float *c, uint n) {
    for (uint i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

int verify_equal(float *a, float *b, uint n)
{
    for (uint i = 0; i < n; i++) {
        if (a[i] != b[i]) return 0;
    }
    return 1;

}


int main() {
    uint n = 1024*1024;

    float *A = (float*)malloc(n * sizeof(float));
    float *B = (float*)malloc(n * sizeof(float));
    float *C = (float*)malloc(n * sizeof(float));


    for(uint i = 0; i < n; i++) {
        A[i] = (float)rand() / RAND_MAX;
        B[i] = (float)rand() / RAND_MAX;
    }

    vecAdd_f32(A, B, C, n);

    float *C_cpu = (float *)malloc(n * sizeof(float));

    add_vectors_cpu(A, B, C_cpu, n);

    printf("Do they match? %s!\n", verify_equal(C, C_cpu, n) ? "Yes" : "No" );

    return 0;
}