#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>

#include "conv2d-helpers.h"

/* 2D convolution, with padding to valid shape. Channel-first */
__global__ void conv2d_pad(float *in,
                           float *out,
                           float *filter,
                           int h,
                           int w,
                           int in_channels,
                           int out_channels,
                           int filter_size /* Must be an odd number */,
                           float pad) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int filter_r = (filter_size - 1) / 2;


    // In and Out data dimensions:
    // 0 - channel
    // 1 - height
    // 2 - width

    // Filter dimensions:
    // 0 - out channels
    // 1 - in channels
    // 2 - height
    // 3 - width

    if (x >= w || y >= h) return;

#ifdef DEBUG
    if (x == 0 && y == 0) PRINT_INPUTS();
#endif

    // Loop over the output channels
    for (int out_c = 0; out_c < out_channels; out_c++) {
        ACCUM_DTYPE R = 0;

        // Pointer to the 2d slice of the output
        float *sub_output = out + out_c * w * h;

        // Loop over the input channels
        for (int in_c = 0; in_c < in_channels; in_c++) {
            // Pointer to the 2d slice of the filter that corresponds to the active input and output
            // channels
            float *sub_filter = filter + (filter_size * filter_size * in_channels * out_c) +
                                (filter_size * filter_size * in_c);
            // Pinter to the current channel in the input
            float *sub_input = in + (w * h * in_c);

            // Apply the filter to the input or the pad value for outside indices.
            for (int filter_y = 0; filter_y < filter_size; filter_y++) {
                for (int filter_x = 0; filter_x < filter_size; filter_x++) {
                    float v = pad;
                    int input_x = x - filter_r + filter_x;
                    int input_y = y - filter_r + filter_y;

                    if (input_x >= 0 && input_x < w && input_y >= 0 && input_y < h) {
                        v = sub_input[input_y * w + input_x];
                    }
                    R += v * sub_filter[filter_y * filter_size + filter_x];
                }
            }
        }
        sub_output[y * w + x] = R;
    }
}
